#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"
#include <math.h>

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<float3, 2> queue_buffer;
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );

// change block_size to how big you want each "pixel" to be
// a value of 2 represents a 2x2 pixel
rtDeclareVariable(const unsigned int, block_size, , ) = 2;
rtDeclareVariable(const unsigned int, half_block_size, , ) = block_size >> 1;


//trace the ray through screen_coord
static __device__ __inline__ float3 trace( float2 screen_coord )
{
  size_t2 screen = output_buffer.size();
  float2 d = screen_coord / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);
  return prd.result;
}

//find the average color of the block of pixels, then find variance of 
//color in the block of pixels and set the pixel color to that color
static __device__ __inline__ void avgColor( const uint2& index )
{
  size_t2 screen = output_buffer.size();         //size of the screen
  float3 color = make_float3(0);                 //keeps track of the colors at each pixel in the block
  float variance_red = 0;						 //variance in the reds
  float variance_green = 0;						 //variance in the greens
  float variance_blue = 0;						 //variance in the blues
  float variance_total = 0;					     //total variance of the colors

  //make sure we look at pixels that are on the screen
  unsigned int min_x = max( index.x-half_block_size, 0u );
  unsigned int max_x = min( index.x+half_block_size, (unsigned int) screen.x );
  unsigned int min_y = max( index.y-half_block_size, 0u );
  unsigned int max_y = min( index.y+half_block_size, (unsigned int) screen.y );
  
  for ( unsigned int i = min_x; i < max_x; ++i ) {
    for ( unsigned int j = min_y; j < max_y; ++j ) {
        //keep a running total of the color values
		queue_buffer[make_uint2(i,j)] = trace(make_float2(i, j));
		color += queue_buffer[make_uint2(i,j)];
    }
  }

  //calculate the average color of the block of pixels 
  color /= (block_size * block_size);

  for ( unsigned int i = min_x; i < max_x; ++i ) {
    for ( unsigned int j = min_y; j < max_y; ++j ) {
	  //variance sum += (sample color - average color)^2      for each sample
	  //calculate variance of each color
	  float3 temp = queue_buffer[make_uint2(i,j)];

	  variance_red += (temp.x - color.x) * (temp.x - color.x);
      variance_green += (temp.y - color.y) * (temp.y - color.y);
      variance_blue += (temp.z - color.z) * (temp.z - color.z);
    }
  }

  //variance = variance sum / num samples
  //standard deviation = sqrtf(variance)
  //multiplied to accentuate colors more
  variance_red = sqrtf((variance_red / (block_size * block_size)))*2;// * 8;
  variance_green = sqrtf((variance_green / (block_size * block_size)))*2;// * 8;
  variance_blue = sqrtf((variance_blue / (block_size * block_size)))*2;// * 8;

  //calculate the luminance value of the three variances
  variance_total = optix::luminance(make_float3(variance_red, variance_green, variance_blue));

  for ( unsigned int i = min_x; i < max_x; ++i ) {
    for ( unsigned int j = min_y; j < max_y; ++j ) {
	  //set the color of all the pixels in the block to the amount of variance among them
	  output_buffer[make_uint2(i, j)] = make_color(make_float3(variance_total));
	}
  }
}


//check whether the pixel is in the center of block
static __device__ __inline__ bool shouldTrace( const uint2& index )
{
  uint2        shifted_index = make_uint2( index.x + half_block_size, index.y + half_block_size ); 
  size_t2      screen        = output_buffer.size(); 
  return ( shifted_index.x % block_size == 0 && shifted_index.y % block_size == 0 ) ||
         ( index.x == screen.x-1 && screen.x % block_size <= half_block_size && shifted_index.y % block_size == 0 ) ||
         ( index.y == screen.y-1 && screen.y % block_size <= half_block_size && shifted_index.x % block_size == 0 );
}


RT_PROGRAM void pinhole_camera()
{
  //check whether or not the current index is at the center of a block
  if(shouldTrace(launch_index)) 
  {
	  avgColor(launch_index);
  }
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color(bad_color);
}